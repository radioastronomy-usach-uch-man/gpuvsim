#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------
  Copyright (C) 2016-2017  Miguel Carcamo, Pablo Roman, Simon Casassus,
  Victor Moral, Fernando Rannou - miguel.carcamo@usach.cl

  This program includes Numerical Recipes (NR) based routines whose
  copyright is held by the NR authors. If NR routines are included,
  you are required to comply with the licensing set forth there.

	Part of the program also relies on an an ANSI C library for multi-stream
	random number generation from the related Prentice-Hall textbook
	Discrete-Event Simulation: A First Course by Steve Park and Larry Leemis,
  for more information please contact leemis@math.wm.edu

  For the original parts of this code, the following license applies:

  This program is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with this program. If not, see <http://www.gnu.org/licenses/>.
* -------------------------------------------------------------------------
*/

#include "functions.cuh"
#include "directioncosines.cuh"
#include <time.h>

long M, N, numVisibilities;
int iter=0;

hipfftHandle plan1GPU;

float2 *device_I;

hipfftComplex *device_I_nu, *device_V;

float beam_noise, beam_bmaj;
float beam_bmin, b_noise_aux, random_probability = 1.0, apply_noise;
float noise_jypix, fg_scale, antenna_diameter, pb_factor, pb_cutoff, nu_0;

dim3 threadsPerBlockNN;
dim3 numBlocksNN;

int threadsVectorReduceNN, blocksVectorReduceNN, verbose_flag = 0, it_maximum, status_mod_in, status_mod_in_alpha;
int selected, t_telescope, reg_term;
char *output;

double ra, dec, DELTAX, DELTAY, deltau, deltav, crpix1, crpix2;

freqData data;
fitsfile *mod_in, *mod_in_alpha;

Field *fields;

VariablesPerField *vars_per_field;

inline bool IsGPUCapableP2P(hipDeviceProp_t *pProp)
{
  #ifdef _WIN32
      return (bool)(pProp->tccDriver ? true : false);
  #else
      return (bool)(pProp->major >= 2);
  #endif
}

inline bool IsAppBuiltAs64()
{
  #if defined(__x86_64) || defined(AMD64) || defined(_M_AMD64)
      return 1;
  #else
      return 0;
  #endif
}

__host__ int main(int argc, char **argv) {
  clock_t t;
  double start, end;
	////CHECK FOR AVAILABLE GPUs
  printf("gpuvsim Copyright (C) 2016-2017  Miguel Carcamo, Pablo Roman, Simon Casassus, Victor Moral, Fernando Rannou - miguel.carcamo@usach.cl\n");
  printf("This program comes with ABSOLUTELY NO WARRANTY; for details use option -w\n");
  printf("This is free software, and you are welcome to redistribute it under certain conditions; use option -c for details.\n\n\n");

	if (!IsAppBuiltAs64()){
        printf("%s is only supported with on 64-bit OSs and the application must be built as a 64-bit target. Test is being waived.\n", argv[0]);
        exit(EXIT_SUCCESS);
  }


	float noise_min = 1E32;

	Vars variables = getOptions(argc, argv);
	char *msinput = variables.input;
	char *msoutput = variables.output;
  char *inputdat = variables.inputdat;
	char *modinput = variables.modin;
  char *alphainput = variables.alpha;
  apply_noise = variables.noise;
  nu_0 = variables.nu_0;
  if(verbose_flag)
    printf("nu_0: %f\n", nu_0);
  selected = variables.select;
  int total_visibilities = 0;
  random_probability = variables.randoms;

  int num_gpus;
  hipGetDeviceCount(&num_gpus);

  if(selected > num_gpus || selected < 0) {
          printf("ERROR. THE SELECTED GPU DOESN'T EXIST\n");
          exit(-1);
  }else{
    hipDeviceProp_t dprop;
    hipGetDeviceProperties(&dprop, selected);
    if(variables.blockSizeX*variables.blockSizeY >= dprop.maxThreadsPerBlock || variables.blockSizeV >= dprop.maxThreadsPerBlock){
        printf("ERROR. The maximum threads per block cannot be greater than %d\n", dprop.maxThreadsPerBlock);
        exit(-1);
    }

    if(variables.blockSizeX >= dprop.maxThreadsDim[0] || variables.blockSizeY >= dprop.maxThreadsDim[1] || variables.blockSizeV >= dprop.maxThreadsDim[0]){
      printf("ERROR. The size of the blocksize cannot exceed X: %d Y: %d Z: %d\n", dprop.maxThreadsDim[0], dprop.maxThreadsDim[1], dprop.maxThreadsDim[2]);
      exit(-1);
    }
  }


  readInputDat(inputdat);
  init_beam(t_telescope);
  if(verbose_flag){
	   printf("Counting data for memory allocation\n");
  }

  canvasVariables canvas_vars = readCanvas(modinput, mod_in, b_noise_aux, status_mod_in, verbose_flag);
  canvasVariables foo = readCanvas(alphainput, mod_in_alpha, b_noise_aux, status_mod_in_alpha, verbose_flag);
  M = canvas_vars.M;
  N = canvas_vars.N;
  DELTAX = canvas_vars.DELTAX;
  DELTAY = canvas_vars.DELTAY;
  ra = canvas_vars.ra;
  dec = canvas_vars.dec;
  crpix1 = canvas_vars.crpix1;
  crpix2 = canvas_vars.crpix2;
  beam_bmaj = canvas_vars.beam_bmaj;
  beam_bmin = canvas_vars.beam_bmin;
  beam_noise = canvas_vars.beam_noise;

  data = countVisibilities(msinput, fields);

  vars_per_field = (VariablesPerField*)malloc(data.nfields*sizeof(VariablesPerField));

  if(verbose_flag){
     printf("Number of fields = %d\n", data.nfields);
	   printf("Number of frequencies = %d\n", data.total_frequencies);
   }

  for(int f=0; f<data.nfields; f++)
  {
  	fields[f].visibilities = (Vis*)malloc(data.total_frequencies*sizeof(Vis));
  	fields[f].device_visibilities = (Vis*)malloc(data.total_frequencies*sizeof(Vis));
  	vars_per_field[f].device_vars = (VPF*)malloc(data.total_frequencies*sizeof(VPF));
  }

  //ALLOCATE MEMORY AND GET TOTAL NUMBER OF VISIBILITIES
  for(int f=0; f<data.nfields; f++){
  	for(int i=0; i < data.total_frequencies; i++){
  		fields[f].visibilities[i].stokes = (int*)malloc(fields[f].numVisibilitiesPerFreq[i]*sizeof(int));
  		fields[f].visibilities[i].uvw = (double3*)malloc(fields[f].numVisibilitiesPerFreq[i]*sizeof(double3));
  		fields[f].visibilities[i].weight = (float*)malloc(fields[f].numVisibilitiesPerFreq[i]*sizeof(float));
  		fields[f].visibilities[i].Vo = (hipfftComplex*)malloc(fields[f].numVisibilitiesPerFreq[i]*sizeof(hipfftComplex));
      fields[f].visibilities[i].Vm = (hipfftComplex*)malloc(fields[f].numVisibilitiesPerFreq[i]*sizeof(hipfftComplex));
      total_visibilities += fields[f].numVisibilitiesPerFreq[i];
  	}
  }

  if(verbose_flag){
	   printf("Reading visibilities and FITS input files...\n");
  }


  readMS(msinput, fields, data);

  if(verbose_flag){
    printf("MS File Successfully Read\n");
    if(beam_noise == -1){
      printf("Beam noise wasn't provided by the user... Calculating...\n");
    }
  }

  //Declaring block size and number of blocks for visibilities
  for(int f=0; f<data.nfields; f++){
  	for(int i=0; i< data.total_frequencies; i++){
  		fields[f].visibilities[i].numVisibilities = fields[f].numVisibilitiesPerFreq[i];
  		long UVpow2 = NearestPowerOf2(fields[f].visibilities[i].numVisibilities);
        fields[f].visibilities[i].threadsPerBlockUV = variables.blockSizeV;
  		fields[f].visibilities[i].numBlocksUV = UVpow2/fields[f].visibilities[i].threadsPerBlockUV;
    }
  }

  hipSetDevice(selected);
  for(int f=0; f<data.nfields; f++){
    for(int i=0; i<data.total_frequencies; i++){
         gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].uvw, sizeof(double3)*fields[f].numVisibilitiesPerFreq[i]));
  		 gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].Vo, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i]));
  		 gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].weight, sizeof(float)*fields[f].numVisibilitiesPerFreq[i]));
         gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].Vm, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i]));
  		 gpuErrchk(hipMalloc(&fields[f].device_visibilities[i].Vr, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i]));
  	}
  }




  hipSetDevice(selected);
  for(int f=0; f<data.nfields; f++){
    gpuErrchk(hipMalloc((void**)&vars_per_field[f].atten_image, sizeof(float)*M*N));
    gpuErrchk(hipMemset(vars_per_field[f].atten_image, 0, sizeof(float)*M*N));
  	for(int i=0; i < data.total_frequencies; i++){

  		gpuErrchk(hipMemcpy(fields[f].device_visibilities[i].uvw, fields[f].visibilities[i].uvw, sizeof(double3)*fields[f].numVisibilitiesPerFreq[i], hipMemcpyHostToDevice));

  		gpuErrchk(hipMemcpy(fields[f].device_visibilities[i].weight, fields[f].visibilities[i].weight, sizeof(float)*fields[f].numVisibilitiesPerFreq[i], hipMemcpyHostToDevice));

  		gpuErrchk(hipMemcpy(fields[f].device_visibilities[i].Vo, fields[f].visibilities[i].Vo, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i], hipMemcpyHostToDevice));

      gpuErrchk(hipMemset(fields[f].device_visibilities[i].Vm, 0, sizeof(hipfftComplex)*fields[f].numVisibilitiesPerFreq[i]));

  	}
  }


  //Declaring block size and number of blocks for Image
  dim3 threads(variables.blockSizeX, variables.blockSizeY);
	dim3 blocks(M/threads.x, N/threads.y);
	threadsPerBlockNN = threads;
	numBlocksNN = blocks;

	noise_jypix = beam_noise / (PI * beam_bmaj * beam_bmin / (4 * log(2) ));

	double deltax = RPDEG_D*DELTAX; //radians
	double deltay = RPDEG_D*DELTAY; //radians
	deltau = 1.0 / (M * deltax);
	deltav = 1.0 / (N * deltay);



	float2 *host_I = (float2*)malloc(M*N*sizeof(float2));
  /////////////////////////////////////////////////////CALCULATE DIRECTION COSINES/////////////////////////////////////////////////
  double raimage = ra * RPDEG_D;
  double decimage = dec * RPDEG_D;
  if(verbose_flag){
    printf("FITS: Ra: %lf, dec: %lf\n", raimage, decimage);
    printf("FITS: Center pix: (%lf,%lf)\n", crpix1-1, crpix2-1);
  }

  double lobs, mobs, lphs, mphs;
  double dcosines_l_pix_ref, dcosines_m_pix_ref, dcosines_l_pix_phs, dcosines_m_pix_phs;

  for(int f=0; f<data.nfields; f++){

      direccos(fields[f].ref_ra, fields[f].ref_dec, raimage, decimage, &lobs,  &mobs);
      direccos(fields[f].phs_ra, fields[f].phs_dec, raimage, decimage, &lphs, &mphs);

      dcosines_l_pix_ref = lobs/-deltax; // Radians to pixels
      dcosines_m_pix_ref = mobs/fabs(deltay); // Radians to pixels

      dcosines_l_pix_phs = lphs/-deltax; // Radians to pixels
      dcosines_m_pix_phs = mphs/fabs(deltay); // Radians to pixels


      if(verbose_flag)
      {
          printf("Ref: l (pix): %e, m (pix): %e\n", dcosines_l_pix_ref, dcosines_m_pix_ref);
          printf("Phase: l (pix): %e, m (pix): %e\n", dcosines_l_pix_phs, dcosines_m_pix_phs);

      }

      fields[f].ref_xobs = (crpix1 - 1.0f) + dcosines_l_pix_ref;// + 6.0f;
      fields[f].ref_yobs = (crpix2 - 1.0f) + dcosines_m_pix_ref;// - 7.0f;

      fields[f].phs_xobs = (crpix1 - 1.0f) + dcosines_l_pix_phs;// + 5.0f;
      fields[f].phs_yobs = (crpix2 - 1.0f) + dcosines_m_pix_phs;// - 7.0f;


      if(verbose_flag) {
          printf("Ref: Field %d - Ra: %.16e (rad), dec: %.16e (rad), x0: %f (pix), y0: %f (pix)\n", f, fields[f].ref_ra, fields[f].ref_dec,
                 fields[f].ref_xobs, fields[f].ref_yobs);
          printf("Phase: Field %d - Ra: %.16e (rad), dec: %.16e (rad), x0: %f (pix), y0: %f (pix)\n", f, fields[f].phs_ra, fields[f].phs_dec,
                 fields[f].phs_xobs, fields[f].phs_yobs);
      }

      if(fields[f].ref_xobs < 0 || fields[f].ref_xobs >= M || fields[f].ref_xobs < 0 || fields[f].ref_yobs >= N) {
          printf("Pointing reference center (%f,%f) is outside the range of the image\n", fields[f].ref_xobs, fields[f].ref_yobs);
          goToError();
      }

      if(fields[f].phs_xobs < 0 || fields[f].phs_xobs >= M || fields[f].phs_xobs < 0 || fields[f].phs_yobs >= N) {
          printf("Pointing phase center (%f,%f) is outside the range of the image\n", fields[f].phs_xobs, fields[f].phs_yobs);
          goToError();
      }
  }
	////////////////////////////////////////////////////////MAKE STARTING IMAGE////////////////////////////////////////////////////////
	float *input_sim;
  float *input_sim_alpha;

  readFITSImageValues(modinput, mod_in, input_sim, status_mod_in, M, N);
  readFITSImageValues(alphainput, mod_in_alpha, input_sim_alpha, status_mod_in_alpha, M, N);


	for(int i=0;i<M;i++){
		for(int j=0;j<N;j++){
			host_I[N*i+j].x = input_sim[N*i+j];
			host_I[N*i+j].y = input_sim_alpha[N*i+j];
		}
	}

  free(input_sim);
  free(input_sim_alpha);
	////////////////////////////////////////////////CUDA MEMORY ALLOCATION FOR DEVICE///////////////////////////////////////////////////


  hipSetDevice(selected);
  gpuErrchk(hipMalloc((void**)&device_V, sizeof(hipfftComplex)*M*N));

  hipSetDevice(selected);

  gpuErrchk(hipMalloc((void**)&device_I, sizeof(float2)*M*N));
  gpuErrchk(hipMemset(device_I, 0, sizeof(float2)*M*N));

  gpuErrchk(hipMalloc((void**)&device_I_nu, sizeof(hipfftComplex)*M*N));
  gpuErrchk(hipMemset(device_I_nu, 0, sizeof(hipfftComplex)*M*N));

  gpuErrchk(hipMemcpy2D(device_I, sizeof(float2), host_I, sizeof(float2), sizeof(float2), M*N, hipMemcpyHostToDevice));

	gpuErrchk(hipMemset(device_V, 0, sizeof(hipfftComplex)*M*N));

  hipSetDevice(selected);
	if ((hipfftPlan2d(&plan1GPU, N, M, HIPFFT_C2C))!= HIPFFT_SUCCESS) {
		printf("cufft plan error\n");
		return -1;
	}

  //Time is taken from first kernel
  t = clock();
  start = omp_get_wtime();
  hipSetDevice(selected);
  for(int f=0; f < data.nfields; f++){
  	for(int i=0; i<data.total_frequencies; i++){
  		hermitianSymmetry<<<fields[f].visibilities[i].numBlocksUV, fields[f].visibilities[i].threadsPerBlockUV>>>(fields[f].device_visibilities[i].uvw, fields[f].device_visibilities[i].Vo, fields[f].visibilities[i].freq, fields[f].numVisibilitiesPerFreq[i]);
  		gpuErrchk(hipDeviceSynchronize());
  	}
  }

  uvsim(device_I);

	//Saving visibilities to disk
  residualsToHost(fields, data, 1, 0);
  printf("Saving residuals to MS...\n");


  if(apply_noise && random_probability < 1.0){
    writeMSSIMSubsampledMC(msinput, msoutput, fields, data, random_probability, apply_noise, verbose_flag);
  }else if(random_probability < 1.0){
    writeMSSIMSubsampled(msinput, msoutput, fields, data, random_probability, verbose_flag);
  }else if(apply_noise){
    writeMSSIMMC(msinput, msoutput, fields, data, apply_noise, verbose_flag);
  }else{
     writeMSSIM(msinput, msoutput, fields, data, verbose_flag);
  }

	printf("Visibilities saved.\n");

	//Free device and host memory
	printf("Free device and host memory\n");
	hipfftDestroy(plan1GPU);
  for(int f=0; f<data.nfields; f++){
  	for(int i=0; i<data.total_frequencies; i++){
  		hipFree(fields[f].device_visibilities[i].uvw);
  		hipFree(fields[f].device_visibilities[i].weight);

  		hipFree(fields[f].device_visibilities[i].Vo);

  		hipfftDestroy(vars_per_field[f].device_vars[i].plan);
  	}
  }

  for(int f=0; f<data.nfields; f++){
  	for(int i=0; i<data.total_frequencies; i++){
      if(fields[f].numVisibilitiesPerFreq[i] != 0){
    		free(fields[f].visibilities[i].uvw);
    		free(fields[f].visibilities[i].weight);
    		free(fields[f].visibilities[i].Vo);
        free(fields[f].visibilities[i].Vm);
      }
  	}
  }

	hipFree(device_I);

	hipFree(device_V);

	free(host_I);
	free(msinput);
	free(msoutput);
	free(modinput);

  fits_close_file(mod_in, &status_mod_in);
  if (status_mod_in) {
    fits_report_error(stderr, status_mod_in);
    goToError();
  }

	return 0;
}
